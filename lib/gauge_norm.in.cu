#include "hip/hip_runtime.h"
#include <gauge_field_order.h>
#include <instantiate.h>
#include <memory>

namespace quda {

  using namespace gauge;

  enum norm_type_ {
    NORM1,
    NORM2,
    ABS_MAX,
    ABS_MIN
  };

  template <typename reg_type, typename real, int Nc, QudaGaugeFieldOrder order>
  double norm(const GaugeField &u, int d, norm_type_ type) {
    double norm_ = 0.0;
    switch(type) {
    case   NORM1: norm_ = FieldOrder<reg_type,Nc,1,order,true,real>(const_cast<GaugeField &>(u)).norm1(d);   break;
    case   NORM2: norm_ = FieldOrder<reg_type,Nc,1,order,true,real>(const_cast<GaugeField &>(u)).norm2(d);   break;
    case ABS_MAX: norm_ = FieldOrder<reg_type,Nc,1,order,true,real>(const_cast<GaugeField &>(u)).abs_max(d); break;
    case ABS_MIN: norm_ = FieldOrder<reg_type,Nc,1,order,true,real>(const_cast<GaugeField &>(u)).abs_min(d); break;
    }
    return norm_;
  }

  template <typename T, bool fixed> struct type_mapper {
    using reg_t = typename mapper<T>::type;
    using store_t = T;
  };

  // fixed-point single-precision field
  template <> struct type_mapper<float, true> {
    using reg_t = float;
    using store_t = int;
  };

  template <typename T, bool fixed, int nColor>
  double norm(const GaugeField &u, int d, norm_type_ type) {
    using reg_t = typename type_mapper<T, fixed>::reg_t;
    using store_t = typename type_mapper<T, fixed>::store_t;
    double norm_ = 0.0;
    switch (u.FieldOrder()) {
    case QUDA_FLOAT2_GAUGE_ORDER: norm_ = norm<reg_t, store_t, nColor, QUDA_FLOAT2_GAUGE_ORDER>(u, d, type); break;
    case QUDA_QDP_GAUGE_ORDER:    norm_ = norm<reg_t, store_t, nColor, QUDA_QDP_GAUGE_ORDER>(u, d, type); break;
    case QUDA_MILC_GAUGE_ORDER:   norm_ = norm<reg_t, store_t, nColor, QUDA_MILC_GAUGE_ORDER>(u, d, type); break;
    default: errorQuda("Gauge field %d order not supported", u.Order());
    }
    return norm_;
  }

  template <int...> struct IntList { };

  template <typename T, bool fixed, int nColor, int...N>
  double norm(const GaugeField &u, int d, norm_type_ type, IntList<nColor, N...>)
  {
    double norm_ = 0.0;
    if (u.Ncolor() == 3 && u.Ncolor() == nColor) {
      norm_ = norm<T, fixed, nColor>(u, d, type);
    } else if (u.Ncolor() > 3 && u.Ncolor() / 2 == nColor) {
      norm_ = norm<T, fixed, 2 * nColor>(u, d, type); // factor of two to account for spin with MG fields
    } else {
      if constexpr (sizeof...(N) > 0) {
        norm_ = norm<T, fixed>(u, d, type, IntList<N...>());
      } else {
        errorQuda("Nc = %d has not been instantiated", u.Ncolor());
      }
    }
    return norm_;
  }

  template <typename T> struct Norm {
    Norm(const GaugeField &u, double &nrm, int d, bool fixed, norm_type_ type)
    {
      if (fixed && u.Precision() > QUDA_SINGLE_PRECISION)
        errorQuda("Fixed point override only enabled for 8-bit, 16-bit and 32-bit fields");

      if (fixed) nrm = norm<T,  true>(u, d, type, IntList<@QUDA_MULTIGRID_NC_NVEC_LIST@>());
      else       nrm = norm<T, false>(u, d, type, IntList<@QUDA_MULTIGRID_NC_NVEC_LIST@>());
    }
  };

  double GaugeField::norm1(int d, bool fixed) const {
    if (reconstruct != QUDA_RECONSTRUCT_NO) errorQuda("Unsupported reconstruct=%d", reconstruct);
    double nrm = 0.0;
    instantiatePrecision<Norm>(*this, nrm, d, fixed, NORM1);
    return nrm;
  }

  double GaugeField::norm2(int d, bool fixed) const {
    if (reconstruct != QUDA_RECONSTRUCT_NO) errorQuda("Unsupported reconstruct=%d", reconstruct);
    double nrm = 0.0;
    instantiatePrecision<Norm>(*this, nrm, d, fixed, NORM2);
    return nrm;
  }

  double GaugeField::abs_max(int d, bool fixed) const {
    if (reconstruct != QUDA_RECONSTRUCT_NO) errorQuda("Unsupported reconstruct=%d", reconstruct);
    double nrm = 0.0;
    instantiatePrecision<Norm>(*this, nrm, d, fixed, ABS_MAX);
    return nrm;
  }

  double GaugeField::abs_min(int d, bool fixed) const {
    if (reconstruct != QUDA_RECONSTRUCT_NO) errorQuda("Unsupported reconstruct=%d", reconstruct);
    double nrm = std::numeric_limits<double>::infinity();
    instantiatePrecision<Norm>(*this, nrm, d, fixed, ABS_MIN);
    return nrm;
  }

  template <class Order> void print_matrix(const Order &o, int d, int parity, unsigned int x_cb)
  {
    for (int r = 0; r < o.Ncolor(); r++) {
      printf("rank %d parity %d x %u row %d", comm_rank(), parity, x_cb, r);
      for (int c = 0; c < o.Ncolor(); c++) {
        auto value = complex<double>(o(d, parity, x_cb, r, c));
        printf(" (%g,%g)", value.real(), value.imag());
      }
      printf("\n");
    }
  }

  template <typename Float, int nColor>
  void genericPrintMatrix(const GaugeField &a, int d, int parity, unsigned int x_cb)
  {
    switch (a.FieldOrder()) {
    case QUDA_FLOAT2_GAUGE_ORDER:
      print_matrix(FieldOrder<double, nColor, 1, QUDA_FLOAT2_GAUGE_ORDER, true, Float>(a), d, parity, x_cb);
      break;
    case QUDA_QDP_GAUGE_ORDER:
      print_matrix(FieldOrder<double, nColor, 1, QUDA_QDP_GAUGE_ORDER, true, Float>(a), d, parity, x_cb);
      break;
    case QUDA_MILC_GAUGE_ORDER:
      print_matrix(FieldOrder<double, nColor, 1, QUDA_MILC_GAUGE_ORDER, true, Float>(a), d, parity, x_cb);
      break;
    default: errorQuda("Unsupported field order %d", a.FieldOrder());
    }
  }

  template <typename Float, int nColor, int... N>
  void genericPrintMatrix(const GaugeField &a, int d, int parity, unsigned int x_cb, IntList<nColor, N...>)
  {
    if (a.Ncolor() == nColor) {
      genericPrintMatrix<Float, nColor>(a, d, parity, x_cb);
    } else {
      if constexpr (sizeof...(N) > 0) {
        genericPrintMatrix<Float, N...>(a, d, parity, x_cb, IntList<N...>());
      } else {
        errorQuda("Not supported Ncolor = %d", a.Ncolor());
      }
    }
  }

  template <typename Float> void genericPrintMatrix(const GaugeField &a, int d, int parity, unsigned int x_cb)
  {
    genericPrintMatrix<Float>(a, d, parity, x_cb, IntList<@QUDA_MULTIGRID_NC_NVEC_LIST @>());
  }

  void genericPrintMatrix(const GaugeField &a, int d, int parity, unsigned int x_cb, int rank)
  {
    if (rank != comm_rank()) return;

    GaugeFieldParam param(a);
    param.field = const_cast<GaugeField *>(&a);
    param.location = QUDA_CPU_FIELD_LOCATION;
    param.create = QUDA_COPY_FIELD_CREATE;
    // if field is a pinned device field then we need to clone it on the host
    bool host_clone
      = (a.Location() == QUDA_CUDA_FIELD_LOCATION && a.MemType() == QUDA_MEMORY_DEVICE && !use_managed_memory()) ? true :
                                                                                                                   false;
    std::unique_ptr<GaugeField> clone_a = !host_clone ? nullptr : std::make_unique<GaugeField>(param);
    const GaugeField &a_ = !host_clone ? a : *clone_a.get();

    switch (a.Precision()) {
    case QUDA_DOUBLE_PRECISION: genericPrintMatrix<double>(a_, d, parity, x_cb); break;
    case QUDA_SINGLE_PRECISION: genericPrintMatrix<float>(a_, d, parity, x_cb); break;
    case QUDA_HALF_PRECISION: genericPrintMatrix<short>(a_, d, parity, x_cb); break;
    case QUDA_QUARTER_PRECISION: genericPrintMatrix<int8_t>(a_, d, parity, x_cb); break;
    default: errorQuda("Precision %d not implemented", a.Precision());
    }
  }

} // namespace quda
